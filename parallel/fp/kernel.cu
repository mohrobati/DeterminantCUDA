#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <windows.h> 
#include <cmath>
#include <iostream>
#include <string.h>
#include <sys/stat.h> 
#include <sys/types.h> 
#include <omp.h>

__global__ void scale(double *a, int size, int index) {
	int i;
	int start = (index*size + index);
	int end = (index*size + size);
	for (i = start + 1; i<end; i++) {
		a[i] = (a[i] / a[start]);
	}
}

__global__ void reduce(double *a, int size, int index) {
	int i;
	int tid = threadIdx.x;
	int start = ((index + tid + 1)*size + index);
	int end = ((index + tid + 1)*size + size);
	for (i = start + 1; i<end; i++) {
		a[i] = a[i] - (a[start] * a[(index*size) + (index + (i - start))]);
	}
}

void gaussianElimination(double* dev_a, int N) {
	int i;
	for (i = 0; i<N; i++) {
		scale << <1, 1 >> >(dev_a, N, i);
		reduce << <1, (N - i - 1) >> >(dev_a, N, i);
	}
}

double calculateDet(double* a, int N) {
	int i, k;
	double* c = (double *) malloc(N*N*sizeof(double));
	double *dev_a, *dev_b, *dev_c;
	double l;
	int threads = ((N*N) - 1);
	hipMalloc((void**)&dev_a, N*N * sizeof(double));
	hipMalloc((void**)&dev_b, N*N * sizeof(double));
	hipMalloc((void**)&dev_c, N*N * sizeof(double));
	hipMemcpy(dev_a, a, N*N * sizeof(double), hipMemcpyHostToDevice);
	gaussianElimination(dev_a, N);
	hipMemcpy(c, dev_a, N*N * sizeof(double), hipMemcpyDeviceToHost);
	double det = 1.0;
	for (i = 0; i<N; i++) {
		for (k = 0; k<N; k++) {
			if (i >= k) {
				l = c[i*N + k];
				if (i == k) {
					det *= l;
				}
			}
			else l = 0;
		}	
	}
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return det;
}

double* readLine(FILE* fpr, int* n) {
	int i, j = 0;
	const int buffSize = 100000;
	char line[buffSize];
	if (fgets(line, buffSize, fpr) == NULL)
		return NULL;
	(*n) = (int) sqrt((strlen(line) + 1)/2);
	double *a = (double *)malloc((*n)*(*n)*sizeof(double));
	for (i = 0; i < strlen(line); i++) {
		char c = line[i];
		if (c >= 48 && c <= 57) {
			a[j] = c - 48.0;
			j++;
		}
	}
	return a;
}

void pipeline(FILE* fpr, FILE* fpw) {
	double* readMat = NULL;
	double* readyForCalculateMat = NULL;
	double calculatedDet, readyForWriteDet;
	int* prevSize = (int*) malloc(sizeof(int));
	int* nextSize = (int*)malloc(sizeof(int));
	int read = 1, calc = 0, write = 0;
	while (read + calc + write > 0) {
	#pragma omp parallel num_threads(3)
		{
		#pragma omp sections
			{
			#pragma omp section
				{
					if (read > 0) {
						readMat = readLine(fpr, nextSize);
						if (readMat == NULL)
							read = 0;
					}
				}
			#pragma omp section
				{
					if (calc > 0) {
						calculatedDet = calculateDet(readyForCalculateMat, *prevSize);
					}
				}
			#pragma omp section
				{
					if (write > 0)
						fprintf(fpw, "%lf\n", readyForWriteDet);
				}
			}
		}
		readyForCalculateMat = readMat;
		*prevSize = *nextSize;
		readyForWriteDet = calculatedDet;
		write = calc;
		calc = read;
	}
}

void readFile(char* dir, char* fileName) {
	FILE *fpr;
	FILE *fpw;
	fpr = fopen(dir, "r+");
	if (fpr == NULL) {
		perror("fopen()");
	}
	char path[100];
	path[0] = '\0';
	strcat(strcat(path, "./data_out/"), fileName);
	fpw = fopen(path, "w");
	if (fpw == NULL) {
		perror("fopen()");
	}
	pipeline(fpr, fpw);
	printf("\n");
	fclose(fpr);
	fclose(fpw);
}

void readDirectory(char* dir) {
	WIN32_FIND_DATA FindFileData;
	HANDLE hFind;
	hFind = FindFirstFile(dir, &FindFileData);
	if (hFind == INVALID_HANDLE_VALUE)
	{
		printf("FindFirstFile failed (%d)\n", GetLastError());
		return;
	}
	FindNextFile(hFind, &FindFileData);
	while (FindNextFile(hFind, &FindFileData) != 0) {
		char path[100];
		path[0] = '\0';
		strcat(strcat(path, "./data_in/"), FindFileData.cFileName);
		readFile(path, FindFileData.cFileName);
	}
}

int main() {
	//mkdir("data_out", 0777);
	#ifndef _OPENMP
		printf("omp is not supported!\n");
		system("pause");
	#endif // !_OPENMP
	double start, end;
	start = omp_get_wtime();
	readDirectory("./data_in/*");
	end = omp_get_wtime();
	printf("Time Elapsed: %lfs\n", end - start);
	system("pause");
	return 0;
}
